
#include <hip/hip_runtime.h>
#include<stdio.h>
#define R 4
#define C 4

__global__ void addMat(int *d){
  int r = threadIdx.x + blockDim.x * blockIdx.x;
  int c = threadIdx.y + blockDim.y * blockIdx.y;

  printf("(%d, %d) \n", r, c);
  if (r<R && c<C){
    d[r*R+c] = 2 * d[r*R+c]; 
  } 
}

int main(){

  int *h;
  int *d;
  int i, j;

  h = (int*)malloc(R*C*sizeof(int));

  for(i=0 ; i < R; i++){
    for(j=0 ; j < C; j++){
      h[i*R+j] = i * j;
    }
  }

  for(i=0 ; i < R; i++){
    for(j=0 ; j < C; j++){
      printf("%d ", h[i*R+j]);
    }
    printf("\n");
  }

  if(hipMalloc(&d, R*C*sizeof(int)) != hipSuccess){
    printf("Cuda Allocation FAiled");
  }
  if(hipMemcpy(d, h, R*C*sizeof(int), hipMemcpyHostToDevice) != hipSuccess){
    printf("Cuda Copying Failed");
  };

  dim3 blocks(2,2);
  dim3 threads(2,2);

  addMat<<<blocks, threads>>>(d);

  if(hipMemcpy(h, d, R*C*sizeof(int), hipMemcpyDeviceToHost) != hipSuccess){
    printf("Cuda Copying Failed here \n");
  };

  for(i=0 ; i < R; i++){
    for(j=0 ; j < C; j++){
      printf("%d ", h[i*R+j]);
    }
    printf("\n");
  }
}