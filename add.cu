
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10000

__global__ void add(float *a, float *w , float *z){
  int i;

  for (i=0 ; i < N; i++){
    a[i] = w[i] + z[i];
  }
}

int main(){
  float *x, *y;
  float *w, *z;
  float *a;
  float *b;

  int i;

  x = (float*)malloc(sizeof(float)*N);
  y = (float*)malloc(sizeof(float)*N);
  b = (float*)malloc(sizeof(float)*N);

  for (i=0 ; i < N; i++)
    x[i] = i;
  
  for (i=0 ; i < N; i++)
    y[i] = 2 * i;

  hipMalloc((float**)&w, sizeof(float)*N);
  hipMemcpy(w, x, sizeof(float)*N, hipMemcpyHostToDevice);

  hipMalloc((float**)&z, sizeof(float)*N);
  hipMemcpy(z, y, sizeof(float)*N, hipMemcpyHostToDevice);

  hipMalloc((float**)&a, sizeof(float)*N);

  add<<<1,1>>>(a, w, z);
  hipMemcpy(b, a, sizeof(float)*N, hipMemcpyDeviceToHost);

  // for (i=0 ; i < N; i++)
  //   printf("b = %f \n", b[i]);

  return 0;

}