
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <stdio.h>
#define DSIZE 5

__global__ void Print(int *A)
{
    printf("Printing from GPU\n");
    for(int i=0;i<DSIZE;i++) 
      printf("d_A[%d]=%d\n",i,A[i]);
}
int main()
{
    int *d_A, *h_A, *h_B;
    int i;

    h_A=(int*)malloc(DSIZE*sizeof(int));
    h_B=(int*)malloc(DSIZE*sizeof(int));

    for(i=0;i<DSIZE;i++) 
      h_A[i]=i;
    
    //copy from host to device
    hipMalloc((int**)&d_A,DSIZE*sizeof(int));
    hipMemcpy(d_A,h_A,DSIZE*sizeof(int), hipMemcpyHostToDevice);

    Print<<<1,1>>>(d_A);

    //copy from device to host
    hipMemcpy(h_B,d_A,DSIZE*sizeof(int),hipMemcpyDeviceToHost);

    printf("Printing from CPU\n");
    for(i=0;i<DSIZE;i++) 
      printf("h_B[%d]=%d\n",i,h_B[i]);
      
    hipFree(d_A); \
    hipDeviceReset();
    return 0;
}